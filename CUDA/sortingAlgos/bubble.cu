#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10

__global__ void bubbleSort(int *d_arr) {
    int tid = threadIdx.x;

    for (int i = 0; i < N; i++) {
        if (tid < N - i - 1) {
            if (d_arr[tid] > d_arr[tid + 1]) {
                int temp = d_arr[tid];
                d_arr[tid] = d_arr[tid + 1];
                d_arr[tid + 1] = temp;
            }
        }
        __syncthreads();  // Synchronize after every pass
    }
}

int main() {
    int h_arr[N] = {5, 9, 2, 7, 1, 3, 8, 6, 0, 4};
    int *d_arr;

    hipMalloc((void**)&d_arr, N * sizeof(int));
    hipMemcpy(d_arr, h_arr, N * sizeof(int), hipMemcpyHostToDevice);

    bubbleSort<<<1, N>>>(d_arr);
    hipDeviceSynchronize();

    hipMemcpy(h_arr, d_arr, N * sizeof(int), hipMemcpyDeviceToHost);

    printf("Sorted array: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_arr[i]);
    }
    printf("\n");

    hipFree(d_arr);
    return 0;
}

